#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <numeric>
#include <string>
#include <fstream>
#include <regex>
#include <stdlib.h>
#include <iostream>
#include <ctime>
#include "Point.h"
#include "Centroid.h"
#include "FileReader.h"

static void CheckCudaErrorAux(const char *, unsigned, const char *,
	hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
* Check the return value of the CUDA runtime API call and exit
* the application if the call has failed.
*/
static void CheckCudaErrorAux(const char *file, unsigned line,
	const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "("
		<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}
void print_matrix(int* matrix, int rows, int cols) {
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			printf("%d ", matrix[i * cols + j]);
		}
		printf("\n");
	}
	printf("\n");
}
void print_matrix(float* matrix, int rows, int cols) {
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			printf("%lf ", matrix[i * cols + j]);
		}
		printf("\n");
	}
	printf("\n");
}
void copy_matrix(float* original, float*copy, int rows, int cols){
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			copy[i*cols + j] = original[i*cols + j];
		}
	}
}
bool hasConverged(int numPtsChanged, int numPts, int toleratedFraction){
	printf("\nsoglia: %d", numPts / toleratedFraction);
	if (numPtsChanged > numPts / toleratedFraction)
		return false;
	return true;

}


__global__ void registerPointsToCentroid_tiling(float* points, float* centroids, int* linkMatrix, int numPoints, int numCentroids, int numAttributes, int centroidTilingSize){
	int numThread = blockIdx.x * blockDim.x + threadIdx.x;

	// ---------------------------------------------------- VERSIONE CON TILING -----------------------------------------------------------------------
	extern __shared__ float sharedCentroids[];
	if (numThread < numPoints){//allora vuol dire che il thread corrisponde ad un Point in memoria
		float minDist = FLT_MAX;
		int closerCentroidID;
		float currentDist;
		for (int i = 0; i < floor(((double)numCentroids / (double)centroidTilingSize)); i++){ //itero per ogni centroide
			//tiling of centroids----------------------------------------------------------------------------------------------
			if (threadIdx.x < centroidTilingSize*numAttributes){
				sharedCentroids[threadIdx.x] = centroids[i*centroidTilingSize*numAttributes + threadIdx.x];
			}
			//end of tiling------------------------------------------------------------------------------------------------------
			for (int k = 0; k < centroidTilingSize; k++){
				__syncthreads();
				currentDist = 0.0;
				for (int j = 0; j < numAttributes; j++){
					currentDist = currentDist + pow(((double)points[numThread*numAttributes + j] - (double)sharedCentroids[k*numAttributes + j]), 2.0);
				}
				if (currentDist < minDist){
					minDist = currentDist;
					closerCentroidID = k + i*centroidTilingSize;
				}
			}


		}
		int index = floor(((double)numCentroids / (double)centroidTilingSize));
		//tiling of remaining centroids - if presents
		if (threadIdx.x < (numCentroids - index*centroidTilingSize)*numAttributes){
			sharedCentroids[threadIdx.x] = centroids[index*centroidTilingSize*numAttributes + threadIdx.x];
		}
		//end of tiling--------------------------
		for (int k = 0; k < numCentroids - centroidTilingSize*index; k++){
			__syncthreads();
			currentDist = 0.0;
			for (int j = 0; j < numAttributes; j++){
				currentDist = currentDist + pow(((double)points[numThread*numAttributes + j] - (double)sharedCentroids[k*numAttributes + j]), 2.0);
			}
			if (currentDist < minDist){
				minDist = currentDist;
				closerCentroidID = centroidTilingSize*index + k;
			}
		}
		linkMatrix[numThread] = closerCentroidID;
	}
}

__global__ void registerPointsToCentroid(float* points, float* centroids, int* linkMatrix, int numPoints, int numCentroids, int numAttributes){
	int numThread = blockIdx.x * blockDim.x + threadIdx.x;
	if (numThread < numPoints){//then the thread is associated to a point in the global memory
		float minDist = FLT_MAX;
		int closerCentroidID;
		float currentDist;
		for (int i = 0; i < numCentroids; i++){ //iterating for all the centroids
			currentDist = 0.0;
			for (int j = 0; j < numAttributes; j++){
				currentDist = currentDist + pow(((double)points[numThread*numAttributes + j] - (double)centroids[i*numAttributes + j]), 2.0);
			}
			if (currentDist < minDist){
				minDist = currentDist;
				closerCentroidID = i;
			}
		} 
		linkMatrix[numThread] = closerCentroidID;
	}
}

int main(){
	std::string pointsPath = "dataset.txt";
	std::string centersPath = "centers.txt";
	std::vector<Point> points{};
	std::vector<Centroid> centroids{};

	int iterationNum = 0;
	int toleratedFraction = 500;
	int numPtsChanged;
	bool convergence = false;

	float *ptsMatrix; //matrice dei punti (numPoints x numAttributes)
	float *centersMatrix; //matrice dei centroidi (numCentroids x numAttributes)
	int *linkMatrix; //matrice che descrive l'appartenenza di punti a centroidi (numCentroids x numPoints)
	int *vectorIndex; //serve per il ricalcolo dei centroidi
	int *oldLinkMatrix; //serve per il calcolo della convergenza

	float *devicePtsMatrix; // versione su device
	float *deviceCentersMatrix; //versione su device
	int *deviceLinkMatrix; //versione su device

	bool tilingEnabled = false;


	int numAttributes;
	int numPoints;
	int numCentroids;
	int centroidTilingSize;

	FileReader* reader = new FileReader();
	(*reader).readFile(pointsPath, &points); //leggo i punti da file
	(*reader).readFile(centersPath, &centroids);

	numAttributes = points.at(0).numAttributes;
	numPoints = points.size();
	numCentroids = centroids.size();

	ptsMatrix = (float*)malloc(numPoints*numAttributes*sizeof(float));
	centersMatrix = (float*)malloc(numCentroids*numAttributes*sizeof(float));
	linkMatrix = (int*)malloc(numPoints*sizeof(int));
	oldLinkMatrix = (int*)malloc(numPoints*sizeof(int));;
 
	if (numCentroids*numAttributes <= 500 ){
		centroidTilingSize = numCentroids;
	}
	else{
		centroidTilingSize = 500;
	}
	

	int start_s = clock(); //registro tempi
	//inizializzo la matrice di punti e il vettore di link
	for (int i = 0; i < numPoints; i++){
		for (int j = 0; j < numAttributes; j++){
			ptsMatrix[i*numAttributes + j] = points.at(i).attributes[j];
		}
		linkMatrix[i] = -1;
	}
	//inizializzo la matrice di centroidi 
	for (int i = 0; i < numCentroids; i++){
		for (int j = 0; j < numAttributes; j++){
			centersMatrix[i*numAttributes + j] = centroids.at(i).attributes[j];
		}
	}
	// alloco la memoria sul device
	CUDA_CHECK_RETURN(
		hipMalloc((void **)&devicePtsMatrix, sizeof(float)* numPoints * numAttributes)
		);
	//copio i Punti  dall'host alla memoria del device
	CUDA_CHECK_RETURN(//poi devo provare a metterli nella constant memory
		hipMemcpy(devicePtsMatrix, ptsMatrix, sizeof(float)* numPoints * numAttributes, hipMemcpyHostToDevice)
		);

	//alloco la memoria necessaria e copio il vettore di link
	CUDA_CHECK_RETURN(
		hipMalloc((void **)&deviceLinkMatrix, sizeof(int)* numPoints)
		);
	CUDA_CHECK_RETURN(
		hipMemcpy(deviceLinkMatrix, linkMatrix, sizeof(int)* numPoints, hipMemcpyHostToDevice)
		);


	//azzero l'indice del numero di punti per centroide
	vectorIndex = (int*)malloc(numCentroids*sizeof(int));
	for (int i = 0; i < numCentroids; i++){
		vectorIndex[i] = 0;
	}
	// ---------------------------------------------------------------------------------------------------------- 
	// ----------------------------------------- inizia ciclo principale ----------------------------------------
	// ----------------------------------------------------------------------------------------------------------
	do{
		printf("\n iterazione esterna numero: %d \n", iterationNum);
		//salve il vettore di link della iterazione precedente
		for (int i = 0; i < numPoints; i++){
			oldLinkMatrix[i] = linkMatrix[i];
		}
		for (int i = 0; i < numCentroids; i++){
			vectorIndex[i] = 0;
		}

		// alloco la memoria sul device
		CUDA_CHECK_RETURN(
			hipMalloc((void **)&deviceCentersMatrix, sizeof(float)* numCentroids * numAttributes)
			);
		//copio i centroidi dall'host alla memoria del device
		CUDA_CHECK_RETURN(
			hipMemcpy(deviceCentersMatrix, centersMatrix, sizeof(float)* numCentroids * numAttributes, hipMemcpyHostToDevice)
			);

		dim3 blockDim(1024, 1);
		dim3 gridDim(ceil(numPoints / 1024.0), 1);

		//print_matrix(centersMatrix, numCentroids, numAttributes);

		//-------------------------------esegue kernel----------------------------------------------------------
		if (!tilingEnabled)
			registerPointsToCentroid << <gridDim, blockDim>> >(devicePtsMatrix, deviceCentersMatrix, deviceLinkMatrix, numPoints, numCentroids, numAttributes);
		
		// -- versione con Tiling --
		else{
			registerPointsToCentroid_tiling << <gridDim, blockDim, centroidTilingSize*numAttributes*sizeof(float) >> >(devicePtsMatrix, deviceCentersMatrix, deviceLinkMatrix, numPoints, numCentroids, numAttributes, centroidTilingSize);
		}

		//
		hipDeviceSynchronize();

		//-------------------------------esegue host------------------------------------------------------------

		//copio dati da device a host
		CUDA_CHECK_RETURN(
			hipMemcpy(linkMatrix, deviceLinkMatrix, numPoints*sizeof(int), hipMemcpyDeviceToHost)
			);

		//svuoto le memorie del device - apparte quelle che mantengo per tutte le iterazioni
		hipFree(deviceCentersMatrix);

		//ricalcolo i centroidi
		//azzero la matrice dei centroidi
		for (int i = 0; i < numCentroids; i++){
			for (int j = 0; j < numAttributes; j++){
				centersMatrix[i*numAttributes + j] = 0;
			}
		}
		numPtsChanged = 0;
		//add position of each point on the right cluster
		int index;
		for (int i = 0; i < numPoints; i++){
			index = linkMatrix[i];
			if (oldLinkMatrix[i] != linkMatrix[i]){//for convergency check
				numPtsChanged++;
			}
			for (int k = 0; k < numAttributes; k++){
				centersMatrix[index*numAttributes + k] += ptsMatrix[i*numAttributes + k];
			}
			vectorIndex[index]++;//tracks the number of points per centroid
		}



		//compute the mean position using vectorIndex
		for (int i = 0; i < numCentroids; i++){
			for (int j = 0; j < numAttributes; j++){
				if (vectorIndex[i]==0)
					centersMatrix[i*numAttributes + j] =-FLT_MAX;
				else
					centersMatrix[i*numAttributes + j] = centersMatrix[i*numAttributes + j] / vectorIndex[i];
			}
		}
		iterationNum++;
		printf("numero punti cambiati: %d", numPtsChanged);
		convergence = hasConverged(numPtsChanged, numPoints, toleratedFraction);
	} while (iterationNum<200 && !convergence);


	int stop_s = clock();
	std::cout << "\ntime: " << (stop_s - start_s) / double(CLOCKS_PER_SEC) << std::endl;

	//print_matrix(centersMatrix, numCentroids, numAttributes);
	hipFree(deviceLinkMatrix);
	hipFree(devicePtsMatrix);
	free(ptsMatrix);
	free(centersMatrix);
	free(linkMatrix);
	free(vectorIndex);
	free(oldLinkMatrix);

	return 0;
}

